#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <stdexcept>
#include <iostream>

#include "../../include/gpu/gpu.h"

hipblasHandle_t hcublas[64];
hiprandGenerator_t random_generator[64];
hipblasStatus_t bstatus;
hiprandStatus_t rstatus;


void check_cuda(hipError_t err,const char *msg)
{
    if(err!=hipSuccess)
    {
        std::string error_type = hipGetErrorString(err);
        std::string text = "[CUDA ERROR]: " + error_type + " ("+ std::to_string(err) + ") raised in " + std::string(msg) + " | (check_cuda)";
        throw std::runtime_error(text);
    }

}
void check_cublas(hipblasStatus_t status, const char *f)
{
    if ( status!=  HIPBLAS_STATUS_SUCCESS)
    {
        std::string text = "error in cublas execution in " + std::string(f) + " | (check_cublas)";
        throw std::runtime_error(text);
    }
}
void hw_info()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i=0;i<nDevices;i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);
        fprintf(stderr,"GPU device %d, %s\n",i,prop.name);
    }
}

void gpu_init()
{

    int nDevices;
    hipGetDeviceCount(&nDevices);

    for (int i=0;i<nDevices;i++)
    {
        hipSetDevice(i);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);

        //fprintf(stderr,"GPU device %d, %s, ready\n",i,prop.name);

        check_cublas(hipblasCreate(&(hcublas[i])),"hipblasCreate");
        //fprintf(stderr,"CuBLAS running on GPU device %s\n",prop.name);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

void gpu_set_device(int device)
{
    hipSetDevice(device);
}

int gpu_devices()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    return nDevices;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

float* gpu_create_tensor(int dev,long int size)
{
    float* devicePointer;
    if (hipSetDevice(dev)!=hipSuccess)
    {
        std::string text = "error setting device "+std::to_string(dev)+" in gpu_create_tensor | (gpu_create_tensor)";
        throw std::runtime_error(text);
    }
    check_cuda(hipMalloc((void**)&devicePointer,size*sizeof(float)),"create_tensor");
    return devicePointer;
}

void gpu_delete_tensor(int dev, float* p)
{
    hipSetDevice(dev);
    check_cuda(hipFree(p),"delete_tensor");
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
void gpu_copy_from(int device, long int size, float *ptr, float *cpu_ptr)
{
    hipSetDevice(device);
    check_cuda(hipMemcpy(cpu_ptr,ptr,size*sizeof(float),hipMemcpyDeviceToHost),"gpu_copy");
}

void gpu_copy_to(int device, long int size, float *cpu_ptr, float *ptr)
{
    hipSetDevice(device);
    //copy to device
    check_cuda(hipMemcpy(ptr,cpu_ptr,size*sizeof(float),hipMemcpyHostToDevice),"gpu_copy");
}

/////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void fill_(float* a, float v, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        a[thread_id_x]=v;
    }
}
void gpu_fill(int device, long int size, float *ptr, float v) {
    hipSetDevice(device);

    setDims(size);

    fill_<<<dimGrid,dimBlock>>>(ptr,v,size);
    check_cuda(hipDeviceSynchronize(),"set");
}

/////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void fill_void_(float* a, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        a[thread_id_x]=thread_id_x;
    }
}
void gpu_fill_void(int device, long int size, float *ptr) {
    hipSetDevice(device);

    setDims(size);

    fill_void_<<<dimGrid,dimBlock>>>(ptr,size);
    check_cuda(hipDeviceSynchronize(),"set");
}


__global__ void print_(float* a, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        printf("%f ",a[thread_id_x]);
    }
}
void gpu_print_(int device, long int size, float *ptr) {
    hipSetDevice(device);

    setDims(size);

    //for(int i=0;i<size;i++) 
      //printf("%f ",ptr[i]);

    //print_<<<dimGrid,dimBlock>>>(ptr,size);
    //check_cuda(hipDeviceSynchronize(),"print");
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void fill_strides_(float* a, float *b, int dim, int *strides,int *nstrides,int *perm,long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        int offset = 0;
        int pos=thread_id_x;
        for (int i=0;i<dim-1;i++){
            offset += (pos/strides[i])*nstrides[perm[i]];
            pos=pos%strides[i];
        }
        offset+=pos*nstrides[perm[dim-1]];
    
        b[offset]=a[thread_id_x];
    }
}
void gpu_permute_(int device, long int size, int dim, int *strides, int *nstrides,int *perm, float *ptr)
{   
    float *ptr2;

    hipSetDevice(device); 
    setDims(size);
    
    // create memory for ptr2
    check_cuda(hipMalloc((void**)&ptr2,size*sizeof(float)),"gpu_contiguous");
    // copy the strides and perm to device memory
    int *strides_d; 
    int *nstrides_d;
    int *perm_d;
    check_cuda(hipMalloc((void**)&strides_d,dim*sizeof(int)),"gpu_contiguous");
    check_cuda(hipMalloc((void**)&nstrides_d,dim*sizeof(int)),"gpu_contiguous");
    check_cuda(hipMalloc((void**)&perm_d,dim*sizeof(int)),"gpu_contiguous");

    check_cuda(hipMemcpy(strides_d,strides,dim*sizeof(int),hipMemcpyHostToDevice),"gpu_contiguous");
    check_cuda(hipMemcpy(nstrides_d,nstrides,dim*sizeof(int),hipMemcpyHostToDevice),"gpu_contiguous");
    check_cuda(hipMemcpy(perm_d,perm,dim*sizeof(int),hipMemcpyHostToDevice),"gpu_contiguous");


    fill_strides_<<<dimGrid,dimBlock>>>(ptr,ptr2,dim,strides_d,nstrides_d,perm_d,size);
    check_cuda(hipDeviceSynchronize(),"gpu_contiguous");

    // copy ptr2 to ptr and delete ptr2
    check_cuda(hipMemcpy(ptr,ptr2,size*sizeof(float),hipMemcpyDeviceToDevice),"gpu_contiguous");
    check_cuda(hipFree(ptr2),"gpu_contiguous");
    check_cuda(hipFree(strides_d),"gpu_contiguous");    
    check_cuda(hipFree(nstrides_d),"gpu_contiguous");
    check_cuda(hipFree(perm_d),"gpu_contiguous");
}
