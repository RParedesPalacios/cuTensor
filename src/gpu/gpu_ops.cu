#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <stdexcept>
#include <iostream>

#include "gpu.h"
#include "gpu_ops.h"

__global__ void gpu_sum_(float* a, float *b, float *c, long int size, bool inc){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        if (inc) 
            c[thread_id_x]+=a[thread_id_x]+b[thread_id_x];
        else 
            c[thread_id_x]=a[thread_id_x]+b[thread_id_x];            
    }
}

void gpu_sum(float *ptrA, float *ptrB, float *ptrC, long int Asize, long int Bsize, int device, bool inc)
{
    hipSetDevice(device);
    setDims(Bsize);

    int m=Asize/Bsize;
    for(int i=0;i<m;i++){
        gpu_sum_<<<dimGrid,dimBlock>>>(ptrA+i*Bsize,ptrB,ptrC+i*Bsize,Bsize,inc);
        check_cuda(hipDeviceSynchronize(),"gpu_sum_");
    }
}

__global__ void gpu_sumf_(float* a, float *c, long int size, float s, int device){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        c[thread_id_x]=a[thread_id_x]+s;
               
    }
}
void gpu_sumf(float *ptrA, float *ptrC, long int size, float s, int device)
{
    hipSetDevice(device);
    setDims(size);
    gpu_sumf_<<<dimGrid,dimBlock>>>(ptrA,ptrC,size,s,device);
    check_cuda(hipDeviceSynchronize(),"gpu_sumf_");
}

// gpu mult2D C=A*B ussin cuBLAS taking into account that the matrices are stored in row-major order
void gpu_mult2D(float *ptrA, float *ptrB, float *ptrC, int m, int n, int k, int device) // m=A0,n=A1,k=B1
{
    hipSetDevice(device);
    
    float alpha = 1.0;
    float beta = 0.0;

    check_cublas(hipblasSgemm(hcublas[device], HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n, &alpha, ptrB, k, ptrA, n, &beta, ptrC, k), "hipblasSgemm");
}

__global__ void gpu_elementwise_product_(float *a, float *b, float *c, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        c[thread_id_x]=a[thread_id_x]*b[thread_id_x];
    }
}   

void gpu_elementwise_product(float *ptrA, float *ptrB, float *ptrC, long int size, int device)
{
    hipSetDevice(device);
    setDims(size);
    gpu_elementwise_product_<<<dimGrid,dimBlock>>>(ptrA,ptrB,ptrC,size);
    check_cuda(hipDeviceSynchronize(),"elementwise_product");
}

// gpu scalar multiplication
__global__ void gpu_mult_(float *a, float *c, long int size, float s){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        c[thread_id_x]=a[thread_id_x]*s;
    }
}

void gpu_mult(float *ptrA, float *ptrC,long int size,float s,int device)
{
    hipSetDevice(device);
    setDims(size);
    gpu_mult_<<<dimGrid,dimBlock>>>(ptrA,ptrC,size,s);
    check_cuda(hipDeviceSynchronize(),"gpu_mult_");
}

__global__ void gpu_inv_(float *a, float *c, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        c[thread_id_x]=1.0/a[thread_id_x];
    }
}

void gpu_inv(float *ptrA, float *ptrC, long int size, int device)
{
    hipSetDevice(device);
    setDims(size);
    
    gpu_inv_<<<dimGrid,dimBlock>>>(ptrA,ptrC,size);    
    check_cuda(hipDeviceSynchronize(),"gpu_inv_");

}


__global__ void gpu_pow_(float *a, float *c, long int size, float s){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        c[thread_id_x]=pow(a[thread_id_x],s);
    }
}

void gpu_pow(float *ptrA, float *ptrC, long int size, float s, int device)
{
    hipSetDevice(device);
    setDims(size);
    gpu_pow_<<<dimGrid,dimBlock>>>(ptrA,ptrC,size,s);
    check_cuda(hipDeviceSynchronize(),"gpu_pow_");
}
