#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <stdexcept>
#include <iostream>

#include "gpu.h"
#include "gpu_ops.h"

__global__ void gpu_sum_(float* a, float *b, float *c, long int size, bool inc){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        if (inc) 
            c[thread_id_x]+=a[thread_id_x]+b[thread_id_x];
        else 
            c[thread_id_x]=a[thread_id_x]+b[thread_id_x];            
    }
}


void gpu_sum(float *ptrA, float *ptrB, float *ptrC, long int Asize, long int Bsize, int device, bool inc)
{
    hipSetDevice(device);
    setDims(Bsize);

    int m=Asize/Bsize;
    for(int i=0;i<m;i++){
        gpu_sum_<<<dimGrid,dimBlock>>>(ptrA+i*Bsize,ptrB,ptrC+i*Bsize,Bsize,inc);
        check_cuda(hipDeviceSynchronize(),"gpu_sum_");
    }
}

// gpu mult2D C=A*B ussin cuBLAS taking into account that the matrices are stored in row-major order
void gpu_mult2D(float *ptrA, float *ptrB, float *ptrC, int m, int n, int k, int device) // m=A0,n=A1,k=B1
{
    hipSetDevice(device);
    
    float alpha = 1.0;
    float beta = 0.0;

    check_cublas(hipblasSgemm(hcublas[device], HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n, &alpha, ptrB, k, ptrA, n, &beta, ptrC, k), "hipblasSgemm");
}