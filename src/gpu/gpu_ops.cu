#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <stdexcept>
#include <iostream>

#include "gpu.h"
#include "gpu_ops.h"

__global__ void gpu_sum_(float* a, float *b, float *c, long int size, bool inc){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        if (inc) 
            c[thread_id_x]+=a[thread_id_x]+b[thread_id_x];
        else 
            c[thread_id_x]=a[thread_id_x]+b[thread_id_x];            
    }
}


void gpu_sum(float *ptrA, float *ptrB, float *ptrC, long int size, int device, bool inc)
{
    hipSetDevice(device);

    setDims(size);

    gpu_sum_<<<dimGrid,dimBlock>>>(ptrA,ptrB,ptrC,size,inc);
    check_cuda(hipDeviceSynchronize(),"gpu_sum_");
}